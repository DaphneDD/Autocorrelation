#include "hip/hip_runtime.h"
//This program calculates the autocorrelation of a given signal stored in signal.txt with CUDA
//Programmer: Xiaoqiong Dong
//Date: June 8, 2019


#include <cmath>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <assert.h>
#include <malloc.h>
#include <stdio.h>
#include <fstream>


//CUDA runtime
#include <hip/hip_runtime.h>

//Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE	64		//number of threads per block
#endif


//how many tries to get an average performance
#ifndef NUMTRIES
#define NUMTRIES 10
#endif

//Setting whether to write out autocorrelation result
#ifndef WRITEAUTOCORR
#define WRITEAUTOCORR 0
#endif

using std::cout;
using std::endl;
using std::cerr;
using std::ofstream;


//function prototypes
float Ranf(float, float);
int Ranf(int, int);
void TimeOfDaySeed();

//calculate autocorrelation (CUDA KERNEL) on the device
__global__ void autocorr(float *array, float *blockSums, int *shift)
{
	__shared__ int blockProducts[BLOCKSIZE];
	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = numItems * wgNum + tnum;
	
	//calculate a single product
	blockProducts[tnum] = array[gid] * array[gid + shift[0]];
	
	//calculate the sum of products in this block
	__syncthreads();
	if (tnum == 0)
	{
		for (int i=1; i<BLOCKSIZE; i++)
			blockProducts[tnum] += blockProducts[i];
		blockSums[wgNum] = blockProducts[tnum];
	}
}

int main(int argc, char *argv[])
{

	int dev = findCudaDevice(argc, (const char**) argv);
	
	//read in data
	FILE *fp = fopen("signal.txt", "r");
	if (fp == NULL)
	{
		cerr << "Cannot open file 'signal.txt'" << endl;
		exit(1);
	}
	int size;
	fscanf(fp, "%d", &size);
	float *h_array = new float[2*size];
	float *h_sums = new float[1*size];
	for (int i=0; i<size; i++)
	{
		fscanf(fp, "%f", &h_array[i]);
		h_array[i + size] = h_array[i];
		h_sums[i] = 0;
	}
	fclose(fp);

	//allocate host memory for storing the sums of products
	float *h_blockSums = new float[size/BLOCKSIZE];  // sums of products returned by each block
	

	//allocate device memory
	float *d_array, *d_blockSums;
	int *d_shift;
	dim3 dims_d_array(size*2, 1, 1);
	dim3 dims_d_blockSums(size/BLOCKSIZE, 1, 1);
	dim3 dims_d_shift(1, 1, 1);
	
	hipError_t status;
	status = hipMalloc(reinterpret_cast<void **>(&d_array), size*2*sizeof(float));
	checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void **>(&d_blockSums), size/BLOCKSIZE*sizeof(float));
	checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void **>(&d_shift), sizeof(int));
	checkCudaErrors(status);
	
	//copy host memory to the device
	status = hipMemcpy(d_array, h_array, size*2*sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);
	
	//setup the execution parameters
	dim3 threads(BLOCKSIZE, 1, 1);
	dim3 grid(size/threads.x, 1, 1);
	
	//create and start timer
	hipDeviceSynchronize();
	
	//allocate CUDA events that we'll use for timing
	hipEvent_t start, stop;
	status = hipEventCreate(&start);
	checkCudaErrors(status);
	status = hipEventCreate(&stop);
	checkCudaErrors(status);
	
	//record the start event
	status = hipEventRecord(start, NULL);
	checkCudaErrors(status);
	
	//execute the kernel
	for (int t=0; t<NUMTRIES; t++)
	{
		for (int shift=0; shift<size; shift++)
		{
			status = hipMemcpy(d_shift, &shift, sizeof(int), hipMemcpyHostToDevice);
			checkCudaErrors(status);
			autocorr <<< grid, threads >>> (d_array, d_blockSums, d_shift);
			
			//copy result from the device to the host
			status = hipMemcpy(h_blockSums, d_blockSums, size/BLOCKSIZE*sizeof(float), hipMemcpyDeviceToHost);
			checkCudaErrors(status);
			
			//calculate the autocorrelation
			for (int j=0; j<size/BLOCKSIZE; j++)
				h_sums[shift] += h_blockSums[j];
		}
	}
	
	//record the stop event
	status = hipEventRecord(stop, NULL);
	checkCudaErrors(status);
	
	//wait for the stop event to complete
	status = hipEventSynchronize(stop);
	checkCudaErrors(status);
	
	float msecTotal = 0.0f;
	status = hipEventElapsedTime(&msecTotal, start, stop);
	checkCudaErrors(status);
	
	//compute and print the performance
	double secondsTotal = 0.001 * static_cast<double>(msecTotal);
	double multsPerSecond = static_cast<double>(size) * static_cast<double>(size) * 2.0 * static_cast<double>(NUMTRIES)/ secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	cout << size << '\t' << BLOCKSIZE << '\t' << megaMultsPerSecond << '\t'; 
	
	//clean up memory;
	delete [] h_array;
	delete [] h_blockSums;
	delete [] h_sums;
	
	status = hipFree(d_array);
	checkCudaErrors(status);
	status = hipFree(d_blockSums);
	checkCudaErrors(status);
	status = hipFree(d_shift);
	checkCudaErrors(status);

	return 0;	
}

